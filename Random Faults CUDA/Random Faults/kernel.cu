#include "hip/hip_runtime.h"
/*
/*
 Random Faults in CUDA
 (C) Bedrich Benes 2020
 bbenes@purdue.edu
*/

#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>
#include <math.h>
#include <vector>			//Standard template library class
#include <GL/freeglut.h>

//in house created libraries
#include "vect3d.h"
#include "trackball.h"
#include "helper.h"         


#pragma comment(lib, "freeglut.lib")

TrackBallC trackball;
bool mouseLeft, mouseMid, mouseRight;

GLint wWindow = 1200;
GLint hWindow = 800;


#define DEBUG
const int MAX = 256;
const int SCENE = 1;
const int maxSteps = 256;

GLint n = 1;
GLfloat a[MAX][MAX];
GLint fill = 1;
#define ELEV 0.0005f

//CUDA stuff
float *d_A;

void Cleanup(bool noError)
{
	hipError_t error;
	// Free device memory
	if (d_A) error = hipFree(d_A);
	if (!noError || error != hipSuccess) printf("Something failed \n");
}



void RandomFaultsCuda();

void Idle(void)
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); //clear all
	glEnable(GL_LIGHT0);
	trackball.Set3DViewCamera();
	glDisable(GL_LIGHTING);
	CoordSyst();
	glEnable(GL_LIGHTING);
	glCallList(SCENE);
	glutSwapBuffers();
}


Vect3d CrossProduct(Vect3d *a, Vect3d *b, Vect3d *cross)
{
	cross->SetX(a->GetY() * b->GetZ() - a->GetZ()* b->GetY());
	cross->SetY(a->GetZ() * b->GetX() - a->GetX()* b->GetZ());
	cross->SetZ(a->GetX() * b->GetY() - a->GetY()* b->GetX());
	return *cross;
}

void Display(void)
{
	int i, j;
	Vect3d v1, v2, v3, v13, v12, n;
	GLfloat materialColor[] = { 0.1f, 0.5f, 0.02f, 1.0f };
	GLfloat materialSpecular[] = { 0,0,0,1 };
	glNewList(SCENE, GL_COMPILE);
	glShadeModel(GL_SMOOTH);
	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE, materialColor);
	glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, materialSpecular);
	if (fill) glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	else glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	for (i = 0; i < MAX - 1; i++)
	{
		glBegin(GL_TRIANGLE_STRIP);
		for (j = 0; j < MAX-1; j++)
		{
//triangle 1
			v1.Set(i / (GLfloat)MAX, j / (GLfloat)MAX, a[i][j]);
			v2.Set((i + 1) / (GLfloat)MAX, j / (GLfloat)MAX, a[i + 1][j]);
			v3.Set((i + 1) / (GLfloat)MAX, (j + 1) / (GLfloat)MAX, a[i + 1][j+1]);
			v12.Set(v1.GetX() - v3.GetX(), v1.GetY() - v3.GetY(), v1.GetZ() - v3.GetZ());
			v13.Set(v1.GetX() - v2.GetX(), v1.GetY() - v2.GetY(), v1.GetZ() - v2.GetZ());
			n.Set(CrossProduct(&v13,&v12,&n));
			n.Normalize();
			glNormal3fv(n);
			glVertex3fv(v1);
			glVertex3fv(v2);
			glVertex3fv(v3);
//triangle 2
			v1.Set(i / (GLfloat)MAX, j / (GLfloat)MAX, a[i][j]);
			v2.Set((i + 1) / (GLfloat)MAX, (j + 1) / (GLfloat)MAX, a[i + 1][j+1]);
			v3.Set((i) / (GLfloat)MAX, (j + 1) / (GLfloat)MAX, a[i][j+1]);
			v12.Set(v1.GetX() - v3.GetX(), v1.GetY() - v3.GetY(), v1.GetZ() - v3.GetZ());
			v13.Set(v1.GetX() - v2.GetX(), v1.GetY() - v2.GetY(), v1.GetZ() - v2.GetZ());
			n.Set(CrossProduct(&v13, &v12, &n));
			n.Normalize();
			glNormal3fv(n);
			glVertex3fv(v1);
			glVertex3fv(v2);
			glVertex3fv(v3);
		}
		glEnd();
	}
	glEndList();
}

void DisplayUgly(void)
{
	int i, j;

	glNewList(SCENE, GL_COMPILE);
	glColor3ub(0, 0, 0);
	if (fill) glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	else glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	for (i = 0; i < MAX - 1; i++)
	{
		glBegin(GL_QUAD_STRIP);
		for (j = 0; j < MAX; j++)
		{
			glColor3f(a[i][j], a[i][j], a[i][j]);
			glVertex3f(i / (GLfloat)MAX, j / (GLfloat)MAX, a[i][j]);
			glVertex3f((i + 1) / (GLfloat)MAX, j / (GLfloat)MAX, a[i + 1][j]);
		}
		glEnd();
	}
	glEndList();
}

void Init(void)
{
	int i, j;

	glClearColor(1.0, 1.0, 1.0, 1.0);
	glClearDepth(1000.f);
	glEnable(GL_DEPTH_TEST);
	for (i = 0; i < MAX; i++)
		for (j = 0; j < MAX; j++)  a[i][j] = 0.5;

}


void myReshape(int w, int h)
{
	glViewport(0, 0, w, h);
	wWindow = w;
	hWindow = h;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(-0.2, 1.2, -0.2, 1.2, -10, 10);
}


void RandomFault(void)
{
	//Write the CPU version here
}


void Key(unsigned char key, GLint i, GLint j)
{
	switch (key)
	{
	case 'f':
	case 'F': fill = (fill == 0); glutPostRedisplay(); break;
	case ' ': //run CPU implementation
	{
		long t1 = clock();
		for (int i = 0; i < maxSteps; i++)
		{
			char name[200];
			sprintf(name, "%i%% done\r", 100 * (i + 1) / maxSteps);
			glutSetWindowTitle(name);
			RandomFault();
		}
		long t2 = clock();
		glutSetWindowTitle("Random Faults in Cuda");
		printf("CPU Running time: %i\n", t2 - t1);
		break;
	}
	case 'c': //run CUDA implementation
	{
		glutSetWindowTitle("Running CUDA");
		long t1 = clock();
		RandomFaultsCuda();
		long t2 = clock();
		glutSetWindowTitle("Random Faults in Cuda");
		printf("CUDA Running time: %i\n", t2 - t1);
		break;
	}
	case 27:
	case 'q':
	case 'Q': exit(0);
	}
	glutPostRedisplay();
}

void Mouse(int button, int state, int x, int y) {
	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
	{
		trackball.Set(true, x, y);
		mouseLeft = true;
	}
	if (button == GLUT_LEFT_BUTTON && state == GLUT_UP)
	{
		trackball.Set(false, x, y);
		mouseLeft = false;
	}
	if (button == GLUT_MIDDLE_BUTTON && state == GLUT_DOWN)
	{
		trackball.Set(true, x, y);
		mouseMid = true;
	}
	if (button == GLUT_MIDDLE_BUTTON && state == GLUT_UP)
	{
		trackball.Set(true, x, y);
		mouseMid = false;
	}
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_DOWN)
	{
		trackball.Set(true, x, y);
		mouseRight = true;
	}
	if (button == GLUT_RIGHT_BUTTON && state == GLUT_UP)
	{
		trackball.Set(true, x, y);
		mouseRight = false;
	}
}

void MouseMotion(int x, int y) {
	if (mouseLeft)  trackball.Rotate(x, y);
	if (mouseMid)   trackball.Translate(x, y);
	if (mouseRight) trackball.Zoom(x, y);
//	glutPostRedisplay();
}

__global__ void RandFaultKernel(float a[MAX][MAX],  //2D array of elements
	const int N, //array is N*N
	const int n) //number of steps to run
{
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	if ((i>=N) || (j>=N)) return;
//Write the kernel here
	a[i][j] += (sin((float)i/ blockDim.x)+ cos((float)j / blockDim.y))*0.001;
}


void RandomFaultsCuda()
{
	hipError_t error;
	int sizeArray;

	//allocate array on the device
	sizeArray = sizeof(float)*MAX*MAX; //2D array of floats
	error = hipMalloc((void**)&d_A, sizeArray);
	//Copy the 2D array from host memory to device memory
	error = hipMemcpy(d_A, a, sizeArray, hipMemcpyHostToDevice);
	if (error != hipSuccess) Cleanup(false);

	//prepare blocks and grid
	const int BLOCKSIZE = 16;
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid(ceil((float)MAX / dimBlock.x),
		         ceil((float)MAX / dimBlock.y));
	// Invoke kernel
	RandFaultKernel << <dimGrid, dimBlock >> > ((float(*)[MAX])d_A, MAX, maxSteps);
	error = hipGetLastError();
	if (error != hipSuccess) printf("Something went wrong: %i\n", error);
	error = hipDeviceSynchronize();
	if (error != hipSuccess) { printf("synchronization is wrong\n"); Cleanup(false); }
	// Copy result from device memory to host memory
	error = hipMemcpy(a, d_A, sizeArray, hipMemcpyDeviceToHost);
	if (error != hipSuccess) { printf("could not copy from device\n"); Cleanup(false); }
	Cleanup(true);
}

// Host code
int main(int argc, char** argv)
{
	srand(5);
	glutInitWindowSize(wWindow, hWindow);
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
	glutCreateWindow("Random Faults");
	Init();
	glutDisplayFunc(Display);
	glutIdleFunc(Idle);
	glutKeyboardFunc(Key);
	glutReshapeFunc(myReshape);
	glutMouseFunc(Mouse);
	glutMotionFunc(MouseMotion);
	glutMainLoop();
	return 0;

}


